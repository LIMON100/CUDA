#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<ctime>
#include<iostream>
#include "hip/hip_runtime.h"
#include ""s

using namespace std;

__global__ void AddInts(int* a, int* b, int count) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < count) {
		a[id] = b[id]; 
	}

}

int main()
{
	srand(time(NULL));

	int count = 100;

	int *h_a = new int[count];
	int *h_b = new int[count];

	for (int i = 0; i < count; i++) {

		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;

	}

	std::cout << "Prior to addition";

	for (int i = 0; i < 5; i++) {
		std::cout << h_a[i] << " " << h_b[i] << std::endl;
	}

	int* d_a, * d_b;

	if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess) {
		std::cout << "Nope";
		return 0;
	}

	if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
		std::cout << "Nope";
		return 0;
	}

	if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "Could not copy" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}


	if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "Could not copy" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	AddInts <<< count / 256 + 1, 256 >> > (d_a, d_b, count);

	if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess) {

		delete[] h_a;
		delete[] h_b;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	for (int i = 0; i < 5; i++) {
		std::cout << "It's a" << h_a[i] << endl;
	}

	hipFree(d_a);
	hipFree(d_b);

	delete[] h_a;
	delete[] h_b;

	return 0;
}