#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>


__global__ void convolution_1d(int* array, int* mask, int* result, int n, int m) {
  
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int r = m / 2;

    int start = tid - r;

    int temp = 0;

    for (int j = 0; j < m; j++) {

        if (((start + j) >= 0) && (start + j < n)) {
            temp += array[start + j] * mask[j];
        }
    }

    result[tid] = temp;
}


void verify_result(int* array, int* mask, int* result, int n, int m) {

    int radius = m / 2;
    int temp;
    int start;

    for (int i = 0; i < n; i++) {

        start = i - radius;
        temp = 0;

        for (int j = 0; j < m; j++) {

            if ((start + j >= 0) && (start + j < n)) {
                temp += array[start + j] * mask[j];
            }
        }
        assert(temp == result[i]);
    }
}



int main() {

    int n = 1 << 20;
    int bytes_n = n * sizeof(int);

    int m = 7;
    int bytes_m = m * sizeof(int);

    std::vector<int> h_array(n);
    std::generate(begin(h_array), end(h_array), []() { return rand() % 100; });


    std::vector<int> h_mask(m);
    std::generate(begin(h_mask), end(h_mask), []() { return rand() % 10; });


    std::vector<int> h_result(n);

    int* d_array, * d_mask, * d_result;
    hipMalloc(&d_array, bytes_n);
    hipMalloc(&d_mask, bytes_m);
    hipMalloc(&d_result, bytes_n);

    hipMemcpy(d_array, h_array.data(), bytes_n, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask.data(), bytes_m, hipMemcpyHostToDevice);

    int THREADS = 256;

    int GRID = (n + THREADS - 1) / THREADS;

    convolution_1d << <GRID, THREADS >> > (d_array, d_mask, d_result, n, m);

    hipMemcpy(h_result.data(), d_result, bytes_n, hipMemcpyDeviceToHost);

    verify_result(h_array.data(), h_mask.data(), h_result.data(), n, m);

    std::cout << "COMPLETED SUCCESSFULLY\n";

    hipFree(d_result);
    hipFree(d_mask);
    hipFree(d_array);

    return 0;
}