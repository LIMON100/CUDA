#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>


__global__ void cub(float* d_out, float* d_in) {

	int id = threadIdx.x;
	float f = d_in[id];
	d_out[id] = f * f * f;

}

int main(int argc , char ** argv)
{
	const int array_size = 64;
	const int bytes = array_size * sizeof(float);

	float h_in[array_size];
	for (int i = 0; i < array_size; i++) {
		h_in[i] = float(i);
	}

	float h_out[array_size];

	float* d_in;
	float* d_out;

	hipMalloc((void**)&d_in, bytes);
	hipMalloc((void**)&d_out, bytes);


	hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

	cub << <1, array >> > (d_out, d_in);

	hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);


	for (int i = 0; i < array_size; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");

	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}