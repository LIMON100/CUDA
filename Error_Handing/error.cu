#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<stdio.h>

__global__ void gpuadd(int *d_a , int *d_b , int *d_c){
  *d_c = *d_a + *d_b;
}

int main()
{

  int h_a , h_b , h_c;
  int *d_a , *d_b , *d_c;

  h_a = 1;
  h_b = 4;

  hipError_t cudaStatus;

  cudaStatus = hipMalloc((void **)&d_c , sizeof(int));

  if(cudaStatus != hipSuccess){
    fprintf(stderr, "Cuda Malloc Failed\n", );
  }

  cudaStatus = hipMalloc((void**)&d_a , sizeof(int));

  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed!");
      goto Error;
  }

  cudaStatus = hipMalloc((void**)&d_b, sizeof(int));
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed!");
      goto Error;
  }

  cudaStatus = hipMemcpy(d_a,&h_a, sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
  }

  cudaStatus = hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
  }


  gpuadd<<<1, 1>>>(d_a, d_b, d_c);


  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error;
  }


  cudaStatus = hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error;
  }
  printf("Passing Parameter by Reference Output: %d + %d = %d\n", h_a, h_b, h_c);

Error:
  hipFree(d_c);
  hipFree(d_a);
  hipFree(d_b);

  return 0;
}
