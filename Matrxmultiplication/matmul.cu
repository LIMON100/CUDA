#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include<hip/hip_runtime.h>



using std::cout;
using std::generate;
using std::vector;

__global__ void MatrixMul(const int* a, const int* b, int* c, int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;

    for (int k = 0; k < N; k++) {

        c[row * N + col] += a[row * N + k] * b[k * N + col];

    }
}

void verify_result(vector<int>& a, vector<int>& b, vector<int>& c, int N) {

    for (int i = 0; i < N; i++) {

        for (int j = 0; j < N; j++) {

            int tmp = 0;
            for (int k = 0; k < N; k++) {

                tmp += a[i * N + k] * b[k * N + j];
            }

            assert(tmp == c[i * N + j]);
        }
    }
}


int main()
{
	return 0;
}