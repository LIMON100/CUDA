#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include<conio.h>
#include<hip/hip_runtime.h>
#include<ctime>
#include<hip/hip_runtime.h>
#include <>

using namespace std;

__global__ void myKernel(unsigned long long* time) {

	__shared__ float shared[1024];

	unsigned long long startTime = clock();

	shared[threadIdx.x]++;

	unsigned long long finishTime = clock();

	*time = (finishTime - startTime);

}

int main()
{
	unsigned long long time;
	unsigned long long* d_time;

	hipMalloc(&d_time, sizeof(unsigned long long));

	for (int i = 0; i < 10; i++) {

		myKernel << <1, 32 >> > (d_time);

		hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);

		std::cout << "Time: " << (time - 14) / 32 << endl;

		std::cout << endl;
	}

	hipFree(d_time);

	_getch();

	hipDeviceReset();

	return 0;
}