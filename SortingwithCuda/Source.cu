#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<hip/hip_runtime.h>

#define arraySize 5
#define threadPerBlock 5

__global__ void addKernel(int* d_a, int* d_b) {

    int count = 0;

    int tid = threadIdx.x;
    int ttid = blockIdx.x * threadPerBlock + tid;


    int val = d_a[ttid];
    __shared__ int cache[threadPerBlock];

    for (int i = tid; i < arraySize; i += threadPerBlock) {
        cache[tid] = d_a[i];
        __syncthreads();

        for (int j = 0; j < threadPerBlock; j++) {
            if (val > cache[j]) {
                count++;
            }
        }
        __syncthreads();
    }
    d_b[count] = val;

}

int main()
{

    int h_a[arraySize] = {3434 , 942, 3234234, 23424, 3438 };
    int h_b[arraySize];
    int* d_a, * d_b;


    hipMalloc((void**)&d_b, arraySize * sizeof(int));
    hipMalloc((void**)&d_a, arraySize * sizeof(int));

    hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice);

    addKernel << <arraySize / threadPerBlock, threadPerBlock >> > (d_a, d_b);


    hipDeviceSynchronize();

    hipMemcpy(h_b, d_b, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    printf("The Enumeration sorted Array is: \n");
    for (int i = 0; i < arraySize; i++) {
        printf("%d\n", h_b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
