#include <stdio.h>
#include <cassert>
#include <iostream>
#include<hip/hip_runtime.h>


#include<hipblas.h>
#include<math.h>

using std::cout;

void vector_init(float* a, int n) {

    for (int i = 0; i < n; i++) {
        a[i] = (float)(rand() % 100);
    }

}

void verify_result(float* a, float* b, float* c, float factor, int n) {

    for (int i = 0; i < n; i++) {
        assert(c[i] == factor * a[i] + b[i]);
    }

}

int main() {
    
    int n = 1 << 2;

    size_t bytes = n * sizeof(float);

    float* h_a, * h_b, * h_c;
    float *d_a, * d_b;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    vector_init(h_a, n);
    vector_init(h_b , n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

    hipblasGetVector(n, sizeof(float) , d_b , 1 , h_c , 1);

    verify_result(h_a, h_b, h_c, scale, n);

    hipblasDestroy(handle);


    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    return 0;
}