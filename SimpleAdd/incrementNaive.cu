#include "hip/hip_runtime.h"
#include<stdio.h>

#define NUM_THREAD 1000000;
#define BLOCK_WIDTH 1000;

#define ARRAY_SIZE 10;


__global__ void incrmnt_nve(int *g){

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  i = i % ARRAY_SIZE;
  g[i] = g[i] + 1;
}


__global__ void incrmnt_atomic(int *g){

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  i = i % ARRAY_SIZE;
  atomicAdd(& g[i] , i);
}


int main(int argc , char **argv){

  int h_array[ARRAY_SIZE];
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

  int *d_array;
  hipMalloc((void **) &d_array , ARRAY_BYTES);
  hipMemset((void *) d_array , 0 , ARRAY_BYTES);

  incrmnt_atomic<<< NUM_THREAD / BLOCK_WIDTH , BLOCK_WIDTH >>>(d_array);

  hipMemcpy(h_array , d_array , ARRAY_BYTES , hipMemcpyDeviceToHost);

  hipFree(d_array);

  return 0;
}
