
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void arrAdd(int *md , int *nd , int *pd , int size){

  int myid = blockIdx.x * blockDim.x + threadIdx.x;
  pd[myid] = md[myid] + nd[myid];

}


int main() {

  int size = 20 * sizeof(int);
  int m[200] , n[200] , p[200] , *md , *nd , *pd;
  int i = 0;

  for(i = 0; i < 200; i++){

    m[i] = i;
    n[i] = i;
    p[i] = 0;

  }

  hipMalloc(&md , size);
  hipMemcpy(md , m , size , hipMemcpyHostToDevice);

  hipMalloc(&nd , size);
  hipMemcpy(nd , n , size , hipMemcpyHostToDevice);

  hipMalloc(&pd , size);

  dim3 DimGrid(10 , 1);
  dim3 DimBlock(200 , 1);

  arrAdd<<< DimGrid , DimBlock >>>(md , nd , pd , size);

  hipMemcpy(p , pd , size , hipMemcpyDeviceToHost);

  hipFree(md);
  hipFree(nd);
  hipFree(pd);

  for(i = 0; i < 200; i++){
    printf("\t%s", p[i]);
  }


  return 0;
}
