#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void global_reduce_kernel(float *d_out , float *d_in) {

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;

  for(unsigned int s = blockDim.x / 2; s > 0; s>>=1){
    if(tid < s){
      d_in[myId] += d_in[myId + s];
    }
    __syncthreads();
  }

  if(tid == 0){
    d_out[blockIdx.x] = d_in[myId]
  }
}


__global__ void shre_reduce_kernel(float *d_out , const float *d_in) {

  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;


  sdata[tid] = d_in[myId];
  __syncthreads();

  for(unsigned int s = blockDim.x / 2; s > 0; s>>=1){
    if(tid < s){
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if(tid == 0){
    d_out[blockIdx.x] = sdata[0];
  }
}


void reduce(float *d_out , float *d_intermediate , float *d_in , int size , bool usesSharedMemory){

  const int maxThreadPerBlock = 1024;
  int threads = maxThreadPerBlock;
  int blocks = size / maxThreadPerBlock;

  if(usesSharedMemory){
    shre_reduce_kernel<<< blocks , threads , threads * sizeof(float) >>>(d_intermediate , d_in);
  }
  else{
    global_reduce_kernel<<< blocks , threads >>>(d_intermediate , d_in);
  }

}

int main(int argc , char **argv){

  int h_array[ARRAY_SIZE];
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

  int *d_array;
  hipMalloc((void **) &d_array , ARRAY_BYTES);
  hipMemset((void *) d_array , 0 , ARRAY_BYTES);

  incrmnt_atomic<<< NUM_THREAD / BLOCK_WIDTH , BLOCK_WIDTH >>>(d_array);

  hipMemcpy(h_array , d_array , ARRAY_BYTES , hipMemcpyDeviceToHost);

  hipFree(d_array);

  return 0;
}
