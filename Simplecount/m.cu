#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void AddInstsCUDA(int* a, int* b)
{
	for (int i = 0; i < 10000005; i++) {
		a[0] += b[0];
	}
}

int main()
{
	int a = 5, b = 9;
	int h_a = 1, h_b = 1;
	int* d_a, * d_b;

	if (hipMalloc((void**)&d_a, sizeof(int)) != cudaSUccess) {
		cout << "Error allocating memory" << endl;
		return 0;
	}

	if (hipMalloc(&d_b, sizeof(int)) != cudaSUccess) {
		cout << "Error allocating memory" << endl;
		free(d_a);
		return 0;
	}

	//hipMalloc(&d_a, sizeof(int));
	//hipMalloc(&d_b, sizeof(int));

	if (hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {

		cout << "ERROR copying memory" << endl;

		CudaFree(d_a);
		CudaFree(d_b);

		return 0;
	}


	if (hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice) != cudaSuceess) {

		cout << "ERROR copying memory" << endl;

		CudaFree(d_a);
		CudaFree(d_b);

		return 0;

	}

	AddInstsCUDA << <1, 1 >> > (d_a, d_b);

	if (hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {

		cout << "ERROR copying memory" << endl;

		CudaFree(d_a);
		CudaFree(d_b);

		return 0;

	}

	cout << "The answer is " << a << endl;

	CudaFree(d_a);
	CudaFree(d_b);

	hipDeviceReset();

	return 0;
}